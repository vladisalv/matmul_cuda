#include "gpu.h"

void Gpu::infoDevices()
{
    int count;
    HANDLE_ERROR( hipGetDeviceCount( &count ) );
    for (int i=0; i< count; i++) {
		printInfoDevice(i);
    }
}

void Gpu::infoMyDevice()
{
	printInfoDevice(myId);
}

void Gpu::setDevice(int major, int minor)
{
	hipDeviceProp_t prop;
	memset(&prop, 0, sizeof(hipDeviceProp_t));

	prop.major = major;
	prop.minor = minor;

	HANDLE_ERROR(hipChooseDevice(&myId, &prop));
	HANDLE_ERROR(hipSetDevice(myId));
}

int Gpu::warpSize()
{
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, myId));
	return prop.warpSize;
}

void Gpu::printInfoDevice(int i)
{
    hipDeviceProp_t  prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
    printf( "   --- General Information for device %d ---\n", i );
    printf( "Name:  %s\n", prop.name );
    printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
    printf( "Clock rate:  %d\n", prop.clockRate );
    printf( "Device copy overlap:  " );
    if (prop.deviceOverlap)
        printf( "Enabled\n" );
    else
        printf( "Disabled\n");
    printf( "Kernel execution timeout :  " );
    if (prop.kernelExecTimeoutEnabled)
        printf( "Enabled\n" );
    else
        printf( "Disabled\n" );

    printf( "   --- Memory Information for device %d ---\n", i );
    printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
    printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
    printf( "Max mem pitch:  %ld\n", prop.memPitch );
    printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

    printf( "   --- MP Information for device %d ---\n", i );
    printf( "Multiprocessor count:  %d\n",
                prop.multiProcessorCount );
    printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
    printf( "Registers per mp:  %d\n", prop.regsPerBlock );
    printf( "Threads in warp:  %d\n", prop.warpSize );
    printf( "Max threads per block:  %d\n",
                prop.maxThreadsPerBlock );
    printf( "Max thread dimensions:  (%d, %d, %d)\n",
                prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                prop.maxThreadsDim[2] );
    printf( "Max grid dimensions:  (%d, %d, %d)\n",
                prop.maxGridSize[0], prop.maxGridSize[1],
                prop.maxGridSize[2] );
    printf( "\n" );
}
